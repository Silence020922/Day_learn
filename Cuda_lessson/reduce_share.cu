#include "hip/hip_runtime.h"
#include <stdio.h>
#include "./tools/setDevice.cuh"

#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif

real reduce_CPU(real *x, const int N){ //CPU中执行规约计算
    real sum = 0.0;
    for (int i = 0;i<N;i++){
        sum += x[i];
    }
    return sum;
}

__global__ void reduce_gpu(real *d_x, real *d_y, const int N){ // 由于每个线程块都会返回一个值，需要存储到数组内
    const int tid = threadIdx.x;
    const int n = blockDim.x*blockIdx.x + threadIdx.x;
    
    __shared__ real s_y[128]; 
    s_y[tid] = (n<N)? d_x[n]:0.0; //对每个线程块的共享内存赋不同的值。
    __syncthreads();

    for (int offset = blockDim.x >> 1;offset > 0;offset >>= 1){ // 使用位操作，相当于 offset /= 2
        if (tid < offset){
            s_y[tid] += s_y[tid + offset];
        }
        __syncthreads(); // 块内同步
    }
    if (tid == 0){
        d_y[blockIdx.x] = s_y[0];
    }
}

void initialData(real *addr, int const elemCount)
{
    for (int i = 0; i < elemCount; i++)
    {
        addr[i] = 1.23; 
    }
    return;
}

int main(){

    SetGPU();
    // 1、分配主机内存，并初始化
    const int iElemCount = 1e8;                     // 设置元素数量
    int blocksize = 128;
    int gridsize = (iElemCount-1)/blocksize + 1;
    size_t stBytesCount = iElemCount * sizeof(real); // 字节数
    
    real *H_x, *H_y;
    H_x = (real *)malloc(stBytesCount); // 分配动态内存
    H_y = (real *)malloc(gridsize * sizeof(real));


    if (H_x != NULL && H_y != NULL)
    {
        memset(H_x, 0, stBytesCount);  // 主机内存初始化为0
        memset(H_y,0,gridsize*sizeof(real));
    }
    else
    {
        printf("Fail to allocate host memory!\n");
        exit(-1);
    }
    real *D_x, *D_y;
    ErrorCheck(hipMalloc((real**)&D_x, stBytesCount), __FILE__, __LINE__);
    ErrorCheck(hipMalloc((real**)&D_y, gridsize*sizeof(real)), __FILE__, __LINE__);
    if (D_x != NULL && D_y != NULL){
    ErrorCheck(hipMemset(D_x, 0, stBytesCount), __FILE__, __LINE__); // 设备内存初始化为0
    ErrorCheck(hipMemset(D_y, 0, gridsize*sizeof(real)), __FILE__, __LINE__);
    }
    else{
        printf("fail to allocate memory\n");
        free(H_x); // 释放先前CPU中制定的内存
        free(H_y);
        exit(-1);
    }

    // 2、初始化主机中数据
    initialData(H_x, iElemCount);
    // 主机复制到设备
    ErrorCheck(hipMemcpy(D_x,H_x,stBytesCount,hipMemcpyHostToDevice),__FILE__,__LINE__);
    ErrorCheck(hipMemcpy(D_y,H_y,gridsize*sizeof(real),hipMemcpyHostToDevice),__FILE__,__LINE__);
    // 3 调用归约函数并记时
    hipEvent_t start, stop; 
    ErrorCheck(hipEventCreate(&start), __FILE__, __LINE__);
    ErrorCheck(hipEventCreate(&stop), __FILE__, __LINE__);
    ErrorCheck(hipEventRecord(start), __FILE__, __LINE__);
    hipEventQuery(start);

    reduce_gpu<<<gridsize,blocksize>>>(D_x,D_y,iElemCount);
    // 将计算结果传回到主机
    ErrorCheck(hipMemcpy(H_y, D_y, gridsize*sizeof(real), hipMemcpyDeviceToHost),__FILE__,__LINE__);
    real sum = reduce_CPU(H_y,gridsize);
    
    ErrorCheck(hipEventRecord(stop), __FILE__, __LINE__);
    ErrorCheck(hipEventSynchronize(stop), __FILE__, __LINE__);
    float elapsed_time;
    ErrorCheck(hipEventElapsedTime(&elapsed_time, start, stop), __FILE__, __LINE__);
    printf("Time = %g ms.\n", elapsed_time); // 打印时间

    // 释放主机与设备内存
    // 4 打印结果
    printf("Sum: %.6f\t\n",sum);
    // 释放主机内存，结束程序
    free(H_x);
    free(H_y);
    ErrorCheck(hipFree(D_x), __FILE__, __LINE__);
    ErrorCheck(hipFree(D_y), __FILE__, __LINE__);
    ErrorCheck(hipDeviceReset(), __FILE__, __LINE__);

    return 0;
}

